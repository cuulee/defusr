#include "hip/hip_runtime.h"
// we get as input two depth feature maps, the lower bound and the reprojected depth, as well as the features
// in addition, we get a culling mask, indicating which of the reprojected values are valid
// we wish to pool them, taking into the culling mask (only for the last two maps)
// The pooling ignores the first element of the second dimension (i.e. the center view)

// input dimensions: (B x N x 1 x H x W), (B x N x 1 x H x W), (B x N x F x H x W), (B x N x 1 x H x W)
// output dimension: (B x 1 x H x W), (B x 1 x H x W), (B x F x H x W)

#define PMRP_SKIP_FIRST (0)
#define MAX_CHANNELS (16)

__global__ void PatchedMaximumReprojectionPooling_forward_kernel(
    float *input_bound,
    float *input_depth,
    float *input_features,
    float *input_mask,
    float *output_bound,
    float *output_depth,
    float *output_features,
    int B, int N, int F, int H, int W
) {
    float max_bound, max_depth;
    float max_feature[MAX_CHANNELS];
    for (int b = blockIdx.x * blockDim.x + threadIdx.x; b < B; b += blockDim.x * gridDim.x) {
    for (int v = blockIdx.y * blockDim.y + threadIdx.y; v < H; v += blockDim.y * gridDim.y) {
    for (int u = blockIdx.z * blockDim.z + threadIdx.z; u < W; u += blockDim.z * gridDim.z) {
        for(int f = 0; f < F; f++) {
            max_feature[f] = 0.0f;
        }
        max_bound = 0.0f;
        max_depth = 0.0f;
        
        int pixel_index = u + v * W;
        for(int n = PMRP_SKIP_FIRST; n < N; n++) {
            int image_index = (b * N + n)* H * W;
            if(max_bound < input_bound[image_index + pixel_index]) {
                max_bound = input_bound[image_index + pixel_index];
            }
            
            if(input_mask[image_index + pixel_index] > 0) {
                if(max_depth < input_depth[image_index + pixel_index]) {
                    max_depth = input_depth[image_index + pixel_index];
                    for(int f = 0; f < F; f++) {
                        max_feature[f] = input_features[image_index * F + f * H * W + pixel_index];
                    }
                }
            }
        }

        output_bound[b * H * W + pixel_index] = max_bound;

        if(max_depth > 0) {
            output_depth[b * H * W + pixel_index] = max_depth;
            
            for(int f = 0; f < F; f++) {
                output_features[(b * F + f) * H * W + pixel_index] = max_feature[f];
            }
        }
    }
    }
    }
}

extern "C" void PatchedMaximumReprojectionPooling_updateOutput_gpu(
    THCudaTensor *input_bound,
    THCudaTensor *input_depth,
    THCudaTensor *input_features,
    THCudaTensor *output_bound,
    THCudaTensor *output_depth,
    THCudaTensor *output_features,
    THCudaTensor *input_mask)
{
    int B = THCudaTensor_size(state, input_features, 0);
    int N = THCudaTensor_size(state, input_features, 1);
    int F = THCudaTensor_size(state, input_features, 2);
    int H = THCudaTensor_size(state, input_features, 3);
    int W = THCudaTensor_size(state, input_features, 4);
 
    const dim3 block = dim3(1, 16, 16);
    const dim3 grid = dim3(1, 4, 4);

    float *input_bound_p     = THCudaTensor_data(state, input_bound);
    float *input_depth_p     = THCudaTensor_data(state, input_depth);
    float *input_features_p  = THCudaTensor_data(state, input_features);
    float *input_mask_p      = THCudaTensor_data(state, input_mask);
    float *output_bound_p    = THCudaTensor_data(state, output_bound);
    float *output_depth_p    = THCudaTensor_data(state, output_depth);
    float *output_features_p = THCudaTensor_data(state, output_features);

    hipStream_t stream = THCState_getCurrentStream(state);
    PatchedMaximumReprojectionPooling_forward_kernel<<<grid, block, 0, stream>>>(
        input_bound_p,
        input_depth_p,
        input_features_p,
        input_mask_p,
        output_bound_p,
        output_depth_p,
        output_features_p,
        B, N, F, H, W
    );

    THCudaCheck(hipGetLastError());
}

__global__ void PatchedMaximumReprojectionPooling_backward_kernel(
    float *input_bound,
    float *input_depth,
    float *dloss_input_bound,
    float *dloss_input_depth,
    float *dloss_input_features,
    float *input_mask,
    float *dloss_output_bound,
    float *dloss_output_depth,
    float *dloss_output_features,
    int B, int N, int F, int H, int W
) {
    float max_bound, max_depth;
    int bound_index, depth_index;
    for (int b = blockIdx.x * blockDim.x + threadIdx.x; b < B; b += blockDim.x * gridDim.x) {
    for (int v = blockIdx.y * blockDim.y + threadIdx.y; v < H; v += blockDim.y * gridDim.y) {
    for (int u = blockIdx.z * blockDim.z + threadIdx.z; u < W; u += blockDim.z * gridDim.z) {
        max_bound = 0.0f;
        max_depth = 0.0f;
        bound_index = 0;
        depth_index = 0;

        // pre-run to get the maximum index, both culled and bound
        int pixel_index = u + v * W;
        for(int n = PMRP_SKIP_FIRST; n < N; n++) {
            int image_index = (b * N + n)* H * W;
            if(max_bound < input_bound[image_index + pixel_index]) {
                max_bound = input_bound[image_index + pixel_index];
                bound_index = n;
            }
            
            if(input_mask[image_index + pixel_index] > 0) {
                if(max_depth < input_depth[image_index + pixel_index]) {
                    max_depth = input_depth[image_index + pixel_index];
                    depth_index = n;
                }
            }
        }
        
        int image_index_bound = (b * N + bound_index)* H * W;
        atomicAdd(
            dloss_input_bound + image_index_bound + pixel_index,
            dloss_output_bound[b * H * W + pixel_index]
        );

        if(max_depth > 0) {
            int image_index_depth = (b * N + depth_index)* H * W;
            atomicAdd(
                dloss_input_depth + image_index_depth + pixel_index,
                dloss_output_depth[b * H * W + pixel_index]
            );
            
            for(int f = 0; f < F; f++) {
                atomicAdd(
                    dloss_input_features + (b * F * N + depth_index * F + f) * H * W + pixel_index,
                    dloss_output_features[(b * F + f) * H * W + pixel_index]
                );
            }
        }
    }
    }
    }
}

extern "C" void PatchedMaximumReprojectionPooling_updateGradInput_gpu(
    THCudaTensor *input_bound,
    THCudaTensor *input_depth,
    THCudaTensor *dloss_input_bound,
    THCudaTensor *dloss_input_depth,
    THCudaTensor *dloss_input_features,
    THCudaTensor *dloss_output_bound,
    THCudaTensor *dloss_output_depth,
    THCudaTensor *dloss_output_features,
    THCudaTensor *input_mask)
{
    int B = THCudaTensor_size(state, dloss_input_features, 0);
    int N = THCudaTensor_size(state, dloss_input_features, 1);
    int F = THCudaTensor_size(state, dloss_input_features, 2);
    int H = THCudaTensor_size(state, dloss_input_features, 3);
    int W = THCudaTensor_size(state, dloss_input_features, 4);
 
    int blkdim = 16;   
    const dim3 block = dim3(1, blkdim, blkdim);
    const dim3 grid = dim3(1, 4, 4);

    float *input_bound_p     = THCudaTensor_data(state, input_bound);
    float *input_depth_p     = THCudaTensor_data(state, input_depth);
    float *dloss_input_bound_p     = THCudaTensor_data(state, dloss_input_bound);
    float *dloss_input_depth_p     = THCudaTensor_data(state, dloss_input_depth);
    float *dloss_input_features_p  = THCudaTensor_data(state, dloss_input_features);
    float *input_mask_p      = THCudaTensor_data(state, input_mask);
    float *dloss_output_bound_p    = THCudaTensor_data(state, dloss_output_bound);
    float *dloss_output_depth_p    = THCudaTensor_data(state, dloss_output_depth);
    float *dloss_output_features_p = THCudaTensor_data(state, dloss_output_features);

    hipStream_t stream = THCState_getCurrentStream(state);
    PatchedMaximumReprojectionPooling_backward_kernel<<<grid, block, 0, stream>>>(
        input_bound_p,
        input_depth_p,
        dloss_input_bound_p,
        dloss_input_depth_p,
        dloss_input_features_p,
        input_mask_p,
        dloss_output_bound_p,
        dloss_output_depth_p,
        dloss_output_features_p,
        B, N, F, H, W
    );

    THCudaCheck(hipGetLastError());
}